#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include ""
using namespace std;

#define blocksize = 16

//Multiplication Kernel 
__global__ void MatMulKernel(int *a, int *b, int *c, int m, int n, int k)
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if ((row < m) && (col < n))
	{
		int pvalue = 0;
		for (int i = 0; i < n; ++i)
		{
			pvalue += a[row*k + i] * b[i*k + col];
		}
		c[row*k + col] = pvalue;
	}
}

int main(int argc, char const *argv)
{
	int m, n, k;
	//fix the seed 
	srand(1);
	cout << "enter m,n,k";
	cin >> m >> n >> k;
	int *h_a, *h_b, *h_c;
	hipHostMalloc((void**)&h_a, sizeof(int)*m*n);
	hipHostMalloc((void**)&h_b, sizeof(int)*m*n);
	hipHostMalloc((void**)&h_c, sizeof(int)*m*n);
	//initlialize a 
	for (int i = 0; i < m; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			h_a[i*n + j] = rand() % 1024;
		}
	}
	for (int i = 0; i < m; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			h_b[i*n + j] = rand() % 1024;
		}
	}
	float gpu_elapsed_time;
	//cuda events to count
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	//allocate deice space 
	int *d_a, *d_b, *d_c;
	hipMalloc((void**)&d_a, sizeof(int)*m*n);
	hipMalloc((void**)&d_b, sizeof(int)*m*n);
	hipMalloc((void**)&d_c, sizeof(int)*m*n);
	//Move data from host to device 
	hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
	hipMemcpy(d_a, h_b, sizeof(int)*m*n, hipMemcpyHostToDevice);
	//Launch the kernel 
	dim3 dimGrid((int)(m / 32), (int)(m / 32), 1);
	dim3 dimBlock(32, 32, 1);
	MatMulKernel << <dimGrid, dimBlock >> > (d_a, d_b, d_c, m, n, k);
	//copy the result back 
	hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	//terminate couting 
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_elapsed_time, start, stop);
	printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time);
	//free the memory 
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);

	return 0;
}
