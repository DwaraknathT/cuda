#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

//kernel 
__global__ vecAdd(float *a, float *b, float c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        c[i] = a[i] + b[i];
    }
}
void vecAdd(float *a, float *b, float *c, int n)
{
    //allocate mem and move data 
    float *d_a, *d_b, *d_c;
    int size = n*sizeof(float);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);

    //move data
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    //launch the kernel 
    vecAdd<<<ceil(n/256.0), 256>>> (d_a, d_b, d_c);
    //copy answer back 
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    //free memeory 
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

